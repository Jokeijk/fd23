#include"field.h"
#include"../gpu.h"
#include"../param/param.h"
#include<cstdio>

/* init fld in CUDA */
void FIELD::init_gpu_full(int deviceid,PARAM & param){
  nx=param.nx;
  nz=param.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1+2*radius;
  safecall(hipMalloc((void**)&(Txx),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(Txz),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(Tzz),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(U  ),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(W  ),sizeof(float)*nx*tnz));
  hipMemset(Txx,  0,sizeof(float)*nx*tnz); Txx  -=(nz1-radius)*nx;
  hipMemset(Txz,  0,sizeof(float)*nx*tnz); Txz  -=(nz1-radius)*nx;
  hipMemset(Tzz,  0,sizeof(float)*nx*tnz); Tzz  -=(nz1-radius)*nx;
  hipMemset(U  ,  0,sizeof(float)*nx*tnz); U    -=(nz1-radius)*nx;
  hipMemset(W  ,  0,sizeof(float)*nx*tnz); W    -=(nz1-radius)*nx;
}

/* load from g_fld to gpu */
void FIELD::cu_load_restart(int deviceid,PARAM &param,FIELD & g_fld)
{
  nx=param.nx;
  nz=param.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1;
  safecall(hipMemcpy(Txx+nz1*nx,g_fld.Txx+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
  safecall(hipMemcpy(Txz+nz1*nx,g_fld.Txz+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
  safecall(hipMemcpy(Tzz+nz1*nx,g_fld.Tzz+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
  safecall(hipMemcpy(U  +nz1*nx,g_fld.U  +nz1*nx,sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
  safecall(hipMemcpy(W  +nz1*nx,g_fld.W  +nz1*nx,sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
}

/* load from g_fld to gpu */
void FIELD::cu_save_state(int deviceid,PARAM &param,FIELD & g_fld)
{
  nx=param.nx;
  nz=param.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1;
  safecall(hipMemcpy(g_fld.Txx+nz1*nx,Txx+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyDeviceToHost));
  safecall(hipMemcpy(g_fld.Txz+nz1*nx,Txz+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyDeviceToHost));
  safecall(hipMemcpy(g_fld.Tzz+nz1*nx,Tzz+nz1*nx,sizeof(float)*nx*tnz,hipMemcpyDeviceToHost));
  safecall(hipMemcpy(g_fld.U  +nz1*nx,U  +nz1*nx,sizeof(float)*nx*tnz,hipMemcpyDeviceToHost));
  safecall(hipMemcpy(g_fld.W  +nz1*nx,W  +nz1*nx,sizeof(float)*nx*tnz,hipMemcpyDeviceToHost));
}
