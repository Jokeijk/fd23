#include"field.h"
#include"../gpu.h"
#include"../param/param.h"
#include<cstdio>

/* init fld in CUDA */
void FIELD::init_gpu_full(int deviceid,PARAM & param){
  nx=param.nx;
  nz=param.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1+2*radius;
  safecall(hipMalloc((void**)&(Txx),sizeof(double)*nx*tnz));
  safecall(hipMalloc((void**)&(Txz),sizeof(double)*nx*tnz));
  safecall(hipMalloc((void**)&(Tzz),sizeof(double)*nx*tnz));
  safecall(hipMalloc((void**)&(U  ),sizeof(double)*nx*tnz));
  safecall(hipMalloc((void**)&(W  ),sizeof(double)*nx*tnz));
  hipMemset(Txx,  0,sizeof(double)*nx*tnz); Txx  -=(nz1-radius)*nx;
  hipMemset(Txz,  0,sizeof(double)*nx*tnz); Txz  -=(nz1-radius)*nx;
  hipMemset(Tzz,  0,sizeof(double)*nx*tnz); Tzz  -=(nz1-radius)*nx;
  hipMemset(U  ,  0,sizeof(double)*nx*tnz); U    -=(nz1-radius)*nx;
  hipMemset(W  ,  0,sizeof(double)*nx*tnz); W    -=(nz1-radius)*nx;
}

/* init from box fld in global to fld in CUDA */
void FIELD::init_gpu_box(int deviceid,PARAM &param){
  if(deviceid != param.boxdevice){
	return;
  }
  nx=param.bw;
  nz=param.bw;
  safecall(hipMalloc((void**)&(Txx),sizeof(double)*nx*nz));
  safecall(hipMalloc((void**)&(Txz),sizeof(double)*nx*nz));
  safecall(hipMalloc((void**)&(Tzz),sizeof(double)*nx*nz));
  safecall(hipMalloc((void**)&(U  ),sizeof(double)*nx*nz));
  safecall(hipMalloc((void**)&(W  ),sizeof(double)*nx*nz));
  hipMemset(Txx,  0,sizeof(double)*nx*nz); 
  hipMemset(Txz,  0,sizeof(double)*nx*nz); 
  hipMemset(Tzz,  0,sizeof(double)*nx*nz); 
  hipMemset(U  ,  0,sizeof(double)*nx*nz); 
  hipMemset(W  ,  0,sizeof(double)*nx*nz); 
}
