#include"material.h"
#include"../param/param.h"
#include"../gpu.h"
#include<cstdio>

void MATERIAL::init_gpu_full(int deviceid,PARAM &param,MATERIAL &mat){
  nx=mat.nx;
  nz=mat.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1;

  safecall(hipMemcpyToSymbol(HIP_SYMBOL("d_coef"),g_coef,sizeof(double)*20));

  usetable=mat.usetable;
  if(usetable){
	num_mat=mat.num_mat;
	usetable=mat.usetable;
	safecall(hipMalloc((void**)&(tbl_BU ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_BW ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MU ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MUA),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_LAM),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(index),sizeof(double)*nx*tnz)); index  -=nz1*nx;

	safecall(hipMemcpy( tbl_BU , mat.tbl_BU , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_BW , mat.tbl_BW , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MU , mat.tbl_MU , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MUA, mat.tbl_MUA, sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_LAM, mat.tbl_LAM, sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( index  + nz1*nx, mat.index  + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));

  }else{
	safecall(hipMalloc((void**)&(BU ),sizeof(double)*nx*tnz)); BU  -=nz1*nx;
	safecall(hipMalloc((void**)&(BW ),sizeof(double)*nx*tnz)); BW  -=nz1*nx;
	safecall(hipMalloc((void**)&(MU ),sizeof(double)*nx*tnz)); MU  -=nz1*nx;
	safecall(hipMalloc((void**)&(MUA),sizeof(double)*nx*tnz)); MUA -=nz1*nx;
	safecall(hipMalloc((void**)&(LAM),sizeof(double)*nx*tnz)); LAM -=nz1*nx;

	safecall(hipMemcpy( BU  + nz1*nx, mat.BU  + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( BW  + nz1*nx, mat.BW  + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MU  + nz1*nx, mat.MU  + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MUA + nz1*nx, mat.MUA + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( LAM + nz1*nx, mat.LAM + nz1*nx, sizeof(double)*nx*tnz,hipMemcpyHostToDevice));

  }

}

void MATERIAL::init_gpu_box(int deviceid,MATERIAL &mat){
  nx=mat.nx;
  nz=mat.nz;
  usetable=mat.usetable;
  if(usetable){
	num_mat=mat.num_mat;
	usetable=mat.usetable;
	safecall(hipMalloc((void**)&(tbl_BU ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_BW ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MU ),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MUA),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(tbl_LAM),sizeof(double)*num_mat));
	safecall(hipMalloc((void**)&(index),sizeof(double)*nx*nz));

	safecall(hipMemcpy( tbl_BU , mat.tbl_BU , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_BW , mat.tbl_BW , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MU , mat.tbl_MU , sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MUA, mat.tbl_MUA, sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_LAM, mat.tbl_LAM, sizeof(double)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( index  , mat.index  , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
  }else{
	safecall(hipMalloc((void**)&(BU ),sizeof(double)*nx*nz));
	safecall(hipMalloc((void**)&(BW ),sizeof(double)*nx*nz));
	safecall(hipMalloc((void**)&(MU ),sizeof(double)*nx*nz));
	safecall(hipMalloc((void**)&(MUA),sizeof(double)*nx*nz));
	safecall(hipMalloc((void**)&(LAM),sizeof(double)*nx*nz));

	safecall(hipMemcpy( BU  , mat.BU  , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( BW  , mat.BW  , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MU  , mat.MU  , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MUA , mat.MUA , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( LAM , mat.LAM , sizeof(double)*nx*nz,hipMemcpyHostToDevice));
  }
}
