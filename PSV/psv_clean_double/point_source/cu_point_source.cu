#include "hip/hip_runtime.h"
#include"point_source.h"
#include"../param/param.h"
#include"../field/field.h"
#include"../gpu.h"
#include"../param/const.h"
void cu_point_source(int deviceid,int it,int lsrc,double *src, PARAM &param, FIELD & fld)
{
  double temp;
  int nx=param.nx;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int zs=param.zs;
  int xs=param.xs;


  double delta=param.dip*PI/180.0;
  double lambda=param.rake*PI/180.0;
  double phi=(param.strike-param.azimuth)*PI/180.0;

  double Mxx = -( sin(delta)*cos(lambda)*sin(2.0*phi) + 
	  sin(2.0*delta)*sin(lambda)*sin(phi)*sin(phi) );

  double Mxz = -( cos(delta)*cos(lambda)*cos(phi) + 
	  cos(2.0*delta)*sin(lambda)*sin(phi) );

  double Mzz = sin(2.0*delta)*sin(lambda);

  int ixs,izs;

  double factor=1.0/PI/1.4142135623731/(param.h*param.h);
  Mxx *= factor;
  Mxz *= factor;
  Mzz *= factor;

  /* Note stress has been normalized by dt,h and so on */


  /* Mxx */
  ixs=xs;  izs=zs;
  if(izs>=nz1 && izs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Txx+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp -= src[it]*Mxx;
	safecall(hipMemcpy(fld.Txx+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }


  /* Mzz */
  ixs=xs;   izs=zs;
  if(zs>=nz1 && zs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Tzz+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp -= src[it]*Mzz;
	safecall(hipMemcpy(fld.Tzz+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }
  /* Mxz */
  for(ixs=xs-1;ixs<=xs;ixs++){
	for(izs=zs;izs<=zs+1;izs++){
	  if(izs>=nz1 && izs<=nz2 && it<lsrc ){ 
		safecall(hipMemcpy(&temp,fld.Txz+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
		temp -= src[it]*Mxz*0.25;
		safecall(hipMemcpy(fld.Txz+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
	  }
	}
  }
}

void cu_point_source_p(int deviceid,int it,int lsrc,double *src, PARAM &param, FIELD & fld)
{
  double temp;
  int nx=param.nx;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int zs=param.zs;
  int xs=param.xs;


  double delta=param.dip*PI/180.0;
  double lambda=param.rake*PI/180.0;
  double phi=(param.strike-param.azimuth)*PI/180.0;

  double Mxx = -( sin(delta)*cos(lambda)*sin(2.0*phi) + 
	  sin(2.0*delta)*sin(lambda)*sin(phi)*sin(phi) );

  double Mxz = -( cos(delta)*cos(lambda)*cos(phi) + 
	  cos(2.0*delta)*sin(lambda)*sin(phi) );

  double Mzz = sin(2.0*delta)*sin(lambda);

  int ixs,izs;

  double factor=1.0/PI/1.4142135623731/(param.h*param.h)*param.dt/param.h;
  Mxx *= factor;
  Mxz *= factor;
  Mzz *= factor;

  /* Note stress has been normalized by dt,h and so on */

  if(it>=lsrc){
	it=lsrc-1;
  }


  /* Mxx */
  ixs=xs;  izs=zs;
  if(izs>=nz1 && izs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Txx+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp -= src[it]*Mxx;
	safecall(hipMemcpy(fld.Txx+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }

  ixs=xs-1;  izs=zs;
  if(izs>=nz1 && izs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Txx+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp += src[it]*Mxx;
	safecall(hipMemcpy(fld.Txx+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }


  /* Mzz */
  ixs=xs;   izs=zs;
  if(zs>=nz1 && zs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Tzz+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp -= src[it]*Mzz;
	safecall(hipMemcpy(fld.Tzz+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }

  ixs=xs-1;   izs=zs;
  if(zs>=nz1 && zs<=nz2 && it<lsrc ){ 
	safecall(hipMemcpy(&temp,fld.Tzz+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
	temp += src[it]*Mzz;
	safecall(hipMemcpy(fld.Tzz+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));
  }
  /* Mxz */
  for(ixs=xs-1;ixs<=xs;ixs++){
	for(izs=zs;izs<=zs+1;izs++){
	  if(izs>=nz1 && izs<=nz2 && it<lsrc ){ 
		safecall(hipMemcpy(&temp,fld.Txz+izs*nx+ixs,sizeof(double),hipMemcpyDeviceToHost));
		temp -= src[it]*Mxz*0.25;
		safecall(hipMemcpy(fld.Txz+izs*nx+ixs,&temp,sizeof(double),hipMemcpyHostToDevice));

		safecall(hipMemcpy(&temp,fld.Txz+izs*nx+ixs-1,sizeof(double),hipMemcpyDeviceToHost));
		temp += src[it]*Mxz*0.25;
		safecall(hipMemcpy(fld.Txz+izs*nx+ixs-1,&temp,sizeof(double),hipMemcpyHostToDevice));
	  }
	}
  }
}
